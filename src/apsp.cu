
#include <hip/hip_runtime.h>
__global__ void floyd_warshall_kernel(const int num_vertices, double *output, double inf, int k) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if ((i < num_vertices) && (j < num_vertices)) {
        int ij = i * num_vertices + j;
        int ik = i * num_vertices + k;
        int kj = k * num_vertices + j;
        if (output[ik] != inf && output[kj] != inf) {
            if (output[ij] > output[ik] + output[kj]) {
                output[ij] = output[ik] + output[kj];
            }
        }
    }
}

void floyd_warshall_gpu(const double *graph, int num_vertices, double *output) {
    int THREADS_PER_BLOCK_SIDE = 16;
    int BLOCKS_PER_GRAPH_SIDE = ((num_vertices + THREADS_PER_BLOCK_SIDE - 1) / THREADS_PER_BLOCK_SIDE);
    dim3 blocks(BLOCKS_PER_GRAPH_SIDE, BLOCKS_PER_GRAPH_SIDE, 1);
    dim3 threadsPerBlock(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE, 1);
    int size = sizeof(double) * num_vertices * num_vertices;

    double *dev_output;
    hipMalloc(&dev_output, size);
    hipMemcpy(dev_output, graph, size, hipMemcpyHostToDevice);

    double inf = std::numeric_limits<double>::max();
    for (int k = 0; k < num_vertices; k++) {
        floyd_warshall_kernel<<<blocks, threadsPerBlock>>>(num_vertices, dev_output, inf, k);
    }

    hipMemcpy(output, dev_output, size, hipMemcpyDeviceToHost);
    hipFree(dev_output);
}
